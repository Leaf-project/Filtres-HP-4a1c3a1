#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_les_gains(Mdl_t * mdl, uint t0, uint t1, uint mode, float GRAND_COEF) {
	ASSERT(GRAND_COEF >= 2);
	ASSERT(mdl->T == (t1-t0));
	float * _y = gpu_vers_cpu<float>(mdl->y__d[C-1], (t1-t0)*P);
	float somme = 0;
	float potentiel = 0;
	FOR(t0, t, t1) {
		somme     += powf(fabs(prixs[t+1]/prixs[t]-1),GRAND_COEF) * (signe((prixs[t+1]/prixs[t]-1)) == signe(_y[(t-t0)*P+0]));
		potentiel += powf(fabs(prixs[t+1]/prixs[t]-1),GRAND_COEF);
	}
	free(_y);
	return somme / potentiel;
};

float mdl_score(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1],    t0, (t1-t0), t_CONTINUE, NULL);
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0), t_CONTINUE, NULL);
	//
	if (mode == 0) return  intel_score_finale(somme_score, (t1-t0), t_CONTINUE, NULL);
	else           return nvidia_score_finale(somme_score, (t1-t0), t_CONTINUE, NULL);
};

float* mdl_pred(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	if (mode == 0) return  intel_prediction(mdl->y[C-1], t0, (t1-t0), t_CONTINUE, NULL);
	else           return nvidia_prediction(mdl->y__d[C-1], t0, (t1-t0), t_CONTINUE, NULL);
};

void mdl_aller_retour(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	//
	float d_score;
	if (mode == 0) d_score =  d_intel_score_finale(somme_score, (t1-t0), _t_MODE, GRAINE);
	else           d_score = d_nvidia_score_finale(somme_score, (t1-t0), _t_MODE, GRAINE);
	//
	if (mode == 0)  d_intel_somme_score(d_score, mdl->y[C-1],    mdl->dy[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	else           d_nvidia_somme_score(d_score, mdl->y__d[C-1], mdl->dy__d[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	mdl_df(mdl, t0, t1, mode, _t_MODE, GRAINE);
};